#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <hipblas.h>
#include <hiprand/hiprand.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))


__global__ void im2col(float *mat, float *col, int K, int channels, int height, int width, int height_col, int width_col, int stride)
{
    
    int tid_j = blockIdx.x*blockDim.x + threadIdx.x;    //column number
    int tid_i = blockIdx.y*blockDim.y + threadIdx.y;    //row number
    int gid = tid_i*(height_col*width_col) + tid_j;    //global_id when reading row major form
    
    if(tid_j < (height_col*width_col))
    {
        int c_im = blockIdx.y;

        int c = gid/(height_col*width_col);//row in which we are working on in the o/p matrix 
        
        int h_offset = (c/K)%K;
        int w_offset = c%K;
        int h =  (gid%(height_col*width_col))/width_col;
        int w = gid%width_col;


        
        int h_pad = h*stride + h_offset;
        int w_pad = w*stride + w_offset;
        
        int index = (c_im * height + h_pad) * width + w_pad;
        
        col[gid] = mat[index];
            
    }
}


__global__ void rearrange_weights(float* wt_mat, float* out_wt_mat, int K, int channels)
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < channels*K*K)
    {
      int row = gid/(K*K);  //the row in the final output matrix that this thread has to work on 
      int off_set = row*(K*K*channels) + row*(K*K) + gid%(K*K); //Exact position where we have to put the value
      out_wt_mat[off_set] = wt_mat[gid];
      
    }
}


void gpuCublasMmul(float *A,  float *B, float *reference,  int m,  int k,  int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,n,m,k,alpha,B,n,A,k,beta,reference,n);    
}

void depth_conv(float *d_mat, float * d_wt_mat, float *out_mat, int stride, int channels, int K, int height, int width, float* im2col_time, float* diag_time, float* cublas_time)
{
    int width_col = (width- K)/stride + 1;
    int height_col = (height - K)/stride + 1;
    size_t totalThreads = channels*K*K*height_col*width_col;            //total elements im2col operation
    size_t dim1 = channels*K*K;                                         //size of weight matrix
    size_t dim2 = channels*channels*K*K;                                //size of output weight matrix
    size_t size = channels*height*width;

    hipError_t error = hipSuccess;
 

    if(d_mat == NULL)
    {
        fprintf(stderr, "depth_convDriver: Input Matrix memory not allocated\n");
        exit(EXIT_FAILURE);       
    }

 
    
    float* d_col = NULL;

    error = hipMalloc((void **)&d_col, totalThreads*sizeof(float));
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: hipMalloc for d_col %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    hipDeviceProp_t devp;
    hipGetDeviceProperties(&devp, 0);

    float num_th = 128.0;
    dim3 gridWeightDim(ceil((channels*K*K)/num_th), 1, 1);
    dim3 blockWeightDim(num_th, 1, 1);
 
      dim3 gridDim(ceil((height_col*width_col)/32.0), channels, 1);
    dim3 blockDim(32, K*K, 1);
 
    
    if (d_wt_mat == NULL)
    {
        fprintf(stderr, "depth_convDriver: No Kernel Paramaters Provided\n");
    }
    float* d_out_wt_mat = NULL;
    error = hipMalloc((void **)&d_out_wt_mat, dim2*sizeof(float));
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: Some Error in hipMalloc for d_out_wt_mat %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    


    hipEvent_t start3, stop3;
    float milliseconds3 = 0;
    hipEventCreate( & start3);
    hipEventCreate( & stop3);
    hipEventRecord(start3);

    rearrange_weights<<<gridWeightDim, blockWeightDim>>>(d_wt_mat, d_out_wt_mat, K, channels);

    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    hipEventElapsedTime( & milliseconds3, start3, stop3);
    printf("Weight Diagonalization: The elapsed time in GPU was %f ms\n", milliseconds3);
    *diag_time = milliseconds3;


    
    
    hipEvent_t start1, stop1;
    float milliseconds1 = 0;
    hipEventCreate( & start1);
    hipEventCreate( & stop1);
    hipEventRecord(start1);

    im2col<<<gridDim ,blockDim>>>(d_mat, d_col, K, channels, height, width, height_col, width_col, stride);
    
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime( & milliseconds1, start1, stop1);
    printf("Im2Col : The elapsed time in GPU was %f ms\n", milliseconds1);
    *im2col_time = milliseconds1;
    
    int nr_rows_A = channels;
    int nr_cols_A = channels*K*K;
    int nr_cols_B = height_col*width_col;
    
    hipEvent_t start2, stop2;
    float milliseconds2 = 0;
    hipEventCreate( & start2);
    hipEventCreate( & stop2);
    hipEventRecord(start2);


    gpuCublasMmul(d_out_wt_mat, d_col, out_mat, nr_rows_A, nr_cols_A, nr_cols_B);

    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime( & milliseconds2, start2, stop2);
    printf("cuBLAS : The elapsed time in GPU was %f ms\n", milliseconds2);
    *cublas_time = milliseconds2;

    
    hipFree(d_col);
    hipFree(d_out_wt_mat);
    
}

int depth_convDriver(float* d_input_mat, float* d_wt_mat, float ** out_mat, int height, int width, int stride, int channels, int K)
{
    
    if (K>5)
    {
        fprintf(stderr, "depth_convDriver: The Kernel Size is greater than 5, may not work\n");
    }

    hipError_t error = hipSuccess;

    float im2_col_total = 0;
    float diag_total = 0;
    float cublas_total = 0;
 
    float im2col_time = 0;
    float diag_time = 0;
    float cublas_time = 0;

    int group_size = 8;//number of channels in a group 
    
    int width_col = (width- K)/stride + 1;
    int height_col = (height - K)/stride + 1;
    

    if (d_wt_mat == NULL)
    {
        fprintf(stderr, "depth_convDriver: No Kernel Paramaters Provided\n");
    }

    if(d_input_mat == NULL)
    {
        fprintf(stderr, "depth_convDriver: Input Matrix memory not allocated\n");
        exit(EXIT_FAILURE);       
    }

    
    float* d_out_mat = NULL;    
    error = hipMalloc((void **)&d_out_mat, channels*width_col*height_col*sizeof(float));        
    if(error != hipSuccess) {
        fprintf(stderr,"depth_convDriver: Error in hipMalloc for Output Matrix (d_out_mat) %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    *out_mat = d_out_mat; //This matrix shall be returned
    
    int input_offset;
    int weight_offset;
    int output_offset;
    int current_channels = group_size;
    
    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate( & start);
    hipEventCreate( & stop);

    hipEventRecord(start);
    printf("Entering\n");

    for(int i = 0; i < channels; i+= group_size)
    {
        input_offset = height*width*i;
      weight_offset = K*K*i;
      output_offset = height_col*width_col*i;
      if ((channels - i) < group_size)
        current_channels = channels - i;

      depth_conv(d_input_mat+input_offset, d_wt_mat+weight_offset, d_out_mat+output_offset , stride, current_channels, K,  height, width, &im2col_time, &diag_time, &cublas_time);
      
      im2_col_total += im2col_time;
      diag_total += diag_time;
      cublas_total += cublas_time;
    }

    printf("Left\n");
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime( & milliseconds, start, stop);


    printf("Depthwise Conv: The elapsed time in GPU was %f ms\n", milliseconds);
    printf("Im2Col: The elapsed time in GPU was %f ms\n", im2_col_total);
    printf("Diagonalwise: The elapsed time in GPU was %f ms\n", diag_total);
    printf("CuBlas GEMM: The elapsed time in GPU was %f ms\n", cublas_total);    
 
    return 0;   
}


int main()
{
    int K, height, width, stride, channels; //kernel size , height of image, width of image, stride, number of channels in the input
    printf("Enter kernel size , height of image, width of image, stride, number of channels in the input\n");
    
    hipError_t error = hipSuccess;

    scanf("%d",&K);
    scanf("%d",&height);
    scanf("%d",&width);
    scanf("%d",&stride);
    scanf("%d",&channels);
    
    int width_col = (width- K)/stride + 1;
    int height_col = (height - K)/stride + 1;
    
    float* wt_mat = (float *)malloc((channels*K*K)*sizeof(float));
    float * d_wt_mat = NULL;
    error = hipMalloc((void **)&d_wt_mat, (channels*K*K)*sizeof(float));        
    if(error != hipSuccess) {
        fprintf(stderr,"depth_conv_example: Error in hipMalloc for Weight Matrix %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    for(int i = 0; i < channels*K*K; i ++)
    {
          wt_mat[i] = 1;
    }
    error = hipMemcpy(d_wt_mat, wt_mat, (channels*K*K)*sizeof(float), hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        fprintf(stderr,"depth_conv_example: Error in copying Weight matrix to Device %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    
 
    size_t size = channels*height*width;
    float* input_mat = (float *)malloc(size*sizeof(float));
    
    float* d_input_mat = NULL;
    error = hipMalloc((void **)&d_input_mat, size*sizeof(float));        
    if(error != hipSuccess) {
        fprintf(stderr,"depth_conv_example: Error in hipMalloc for Input Matrix %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i < size; i++)
    {
            input_mat[i] = 1;
    }
 
    error = hipMemcpy(d_input_mat, input_mat, size*sizeof(float), hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        fprintf(stderr,"depth_conv_example: Error in copying input matrix to Device %s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


    
    float* out_mat = (float *)malloc(channels*height_col*width_col*sizeof(float));
    if(out_mat == NULL)
    {
        fprintf(stderr, "depth_conv_example: Unable to allocate host Output Memory (out_mat)\n");
        exit(EXIT_FAILURE);
    }

    float* d_out_mat = NULL;


    depth_convDriver(d_input_mat,d_wt_mat,&d_out_mat,height,width,stride, channels,K);

    error = hipMemcpy(out_mat,d_out_mat,channels*height_col*width_col*sizeof(float), hipMemcpyDeviceToHost);
    if(error != hipSuccess) {
        fprintf(stderr,"depth_conv_example: Error in copying Output matrix from Device to host%s\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }    
    

    

    free(wt_mat);
    free(input_mat);
    free(out_mat);
    hipFree(d_input_mat);
    hipFree(d_out_mat);
    hipFree(d_wt_mat);
    
}